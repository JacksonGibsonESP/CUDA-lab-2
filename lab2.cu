#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

using namespace std;

#define MIN(a,b) (((a)<(b))?(a):(b))
#define MAX(a,b) (((a)>(b))?(a):(b))
#define LIMIT(a,b,c) (MIN(MAX((a),(b)),(c)))

#define RED		0
#define GREEN	1
#define BLUE	2

#define CSC(call) {														\
    hipError_t err = call;												\
    if(err != hipSuccess) {											\
        fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n",	\
            __FILE__, __LINE__, hipGetErrorString(err));				\
        exit(1);														\
			    }														\
} while (0)

__global__ void kernel_main(int height, int width, int r, unsigned int *src, unsigned int *dst)
{
	int tid_y = threadIdx.y + blockIdx.y * blockDim.y;
	while (tid_y < height)
	{
		int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
		while (tid_x < width)
		{
			int arr_size;
			arr_size = (MIN(tid_y + r, height - 1) - MAX(tid_y - r, 0) + 1) * (MIN(tid_x + r, width - 1) - MAX(tid_x - r, 0) + 1);

			int median = arr_size / 2 + 1;

			unsigned int answer = 0;

			for (int j = 0; j < 3; j++)
			{
				unsigned short int C[256];
				memset(&C, 0, sizeof(unsigned short int) * 256);

				int y_up = MAX(tid_y - r, 0);
				int y_down = MIN(tid_y + r, height - 1);

				int x_left = MAX(tid_x - r, 0);
				int x_right = MIN(tid_x + r, width - 1);

				for (int k = y_up; k <= y_down; k++)
				{
					for (int l = x_left; l <= x_right; l++)
					{
						C[(src[k * width + l] >> (8 * j)) & 0xFF]++;
					}
				}

				if (C[0] < median)
				{
					for (int i = 1; i < 256; i++)
					{
						C[i] += C[i - 1];
						if (C[i] >= median)
						{
							answer |= i << (8 * j);
							break;
						}
					}
				}
			}
			dst[tid_y * width + tid_x] = answer;
			tid_x += blockDim.x * gridDim.x;
		}
		tid_y += blockDim.y * gridDim.y;
	}
}

int main()
{
	string path_in, path_out;

	cin >> path_in >> path_out;

	int r;
	cin >> r;

	int width, height;
	FILE *in = fopen(path_in.c_str(), "rb");
	if (in == NULL)
	{
		cout << "ERROR: Incorrect input file.\n";
		return 0;
	}
	fread(&width, sizeof(int), 1, in);
	fread(&height, sizeof(int), 1, in);

	if (width <= 0 || height <= 0 || r < 0 || r > 100)
	{
		cout << "ERROR: Incorrect data.\n";
		return 0;
	}

	unsigned int *src = (unsigned int *)malloc(sizeof(unsigned int) * width * height);
	unsigned int *dst = (unsigned int *)malloc(sizeof(unsigned int) * width * height);
	fread(src, sizeof(unsigned int), width * height, in);
	fclose(in);

	unsigned int *src_dev;
	CSC(hipMalloc(&src_dev, sizeof(unsigned int) * height * width));
	CSC(hipMemcpy(src_dev, src, sizeof(unsigned int) * height * width, hipMemcpyHostToDevice));

	free(src);

	unsigned int *dst_dev;
	CSC(hipMalloc(&dst_dev, sizeof(unsigned int) * height * width));

	dim3 threads_count(16, 16);

	unsigned int blocks_count_x = LIMIT(1, width  / threads_count.x + 1, 32);
	unsigned int blocks_count_y = LIMIT(1, height / threads_count.y + 1, 32);
	dim3 blocks_count(blocks_count_x, blocks_count_y);

	/*hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);*/

	kernel_main << < blocks_count, threads_count >> >(height, width, r, src_dev, dst_dev);

	/*hipEventRecord(stop);
	hipEventSynchronize(stop);*/

	CSC(hipFree(src_dev));
	CSC(hipMemcpy(dst, dst_dev, sizeof(unsigned int) * height * width, hipMemcpyDeviceToHost));
	CSC(hipFree(dst_dev));

	FILE *out = fopen(path_out.c_str(), "wb");
	if (out == NULL)
	{
		cout << "ERROR: Incorrect output file.\n";
		return 0;
	}
	fwrite(&width, sizeof(int), 1, out);
	fwrite(&height, sizeof(int), 1, out);
	fwrite(dst, sizeof(unsigned int), height * width, out);
	fclose(out);

	free(dst);

	/*float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	cout << milliseconds << "\n";*/
	//hipProfilerStop();
	return 0;
}